
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<iostream>
//https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
__global__ void staticReverse(int *a,int *d, int n)
{
//   __shared__ int s[64];
  int t = threadIdx.x;
  int tr = n-t-1;
//    s[t] = a[t];
  d[t] = a[tr];
  __syncthreads();
 
}

__global__ void dynamicReverse(int *d, int n)
{
  extern __shared__ int s[];
  int t = threadIdx.x;
  int tr = n-t-1;
  s[t] = d[t];
  __syncthreads();
  d[t] = s[tr];
}

int main(void)
{
  const int n = 64;
  int *a, *r, *d;
  
 

  
  
  
  // run version with static shared memory
  hipMallocManaged(&a, n * sizeof(int));
  hipMallocManaged(&d, n * sizeof(int));
  hipMallocManaged(&r, n * sizeof(int));
//   cudaMallocManaged(&d_d, n * sizeof(int));


 for (int i = 0; i < n; i++) {
    a[i] = i;
    r[i] = n-i-1;
    d[i] = 0;
  }


  staticReverse<<<1,n>>>(a,d, n);
  hipDeviceSynchronize();



  for (int i = 0; i < n; i++) {
std::cout << a[i] << " " << d[i] << std::endl;
  //  if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);
 


}
  // run dynamic shared memory version
std::cout << "new" << std::endl;
  dynamicReverse<<<1,n,n*sizeof(int)>>>(d, n);
  hipDeviceSynchronize();
  
  
  
  for (int i = 0; i < n; i++){ 
std::cout << d[i] << " " << r[i]<< std::endl;

    if (d[i] != r[i]) printf("Error: d[%d]!=r[%d] (%d, %d)\n", i, i, d[i], r[i]);


  }
}
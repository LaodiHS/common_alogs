#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <stdio.h>


__global__ void init_primes_kernel(int *prime, unsigned int n)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	while(index + offset < n){
		prime[index + offset] = index + offset + 1;

		offset += stride;
	}
}


__global__ void sieve_of_eratosthenes_kernel(int *prime, unsigned int n, unsigned int sqrRootN)
{
	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x + 2;
	unsigned int stride = gridDim.x*blockDim.x;
	unsigned int offset = 0;

	while(index + offset < sqrRootN){
		unsigned int temp = index + offset;
		for(unsigned int i=temp*temp;i<n;i+=temp){
			prime[i-1] = 0;
		}

		offset += stride;
	}

}


// __global__ void sieve_of_eratosthenes_kernel2(int *prime, unsigned int n)
// {
// 	unsigned int index = threadIdx.x + blockIdx.x*blockDim.x;
// 	unsigned int stride = gridDim.x*blockDim.x;
// 	unsigned int offset = 0;

// 	__shared__ int cache[10];

// 	while(index+offset < n){
// 		cache[threadIdx.x] = prime[index+offset];

// 		__syncthreads();

// 		unsigned int temp = threadIdx.x + 2;
// 		unsigned int start = temp*temp;
// 		unsigned int finish = blockIdx.x*blockDim.x + blockDim.x;
// 		for(unsigned int i=start;i<finish;i+=temp){
// 			if(cache[threadIdx.x] % i == 0){
// 				cache[threadIdx.x] = 0;
// 			}
// 		}

// 		// unsigned int temp = threadIdx.x + 2;
// 		// unsigned int start = max(temp*temp, index+offset) % blockDim.x;
// 		// unsigned int finish = blockDim.x;//(blockIdx.x*blockDim.x + blockDim.x) / blockDim.x;

// 		// if(threadIdx.x == 1){
// 		// 	printf("%d  %d\n", start, finish);
// 		// }

// 		// for(unsigned int i=start;i<finish;i+=temp){
// 		// 	cache[i-1] = 0;
// 		// }

// 		prime[index+offset] = cache[threadIdx.x];

// 		offset += stride;
// 	}
// }
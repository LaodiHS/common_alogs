#include <iostream>
#include <time.h>
#include <random>
#include "kernels.cuh"

using namespace std;



int main()
{
	unsigned int n = 16;

	int *h_x;
	int *d_x;
	int *h_root;
	int *d_root;
	int *h_child;
	int *d_child;

	// allocate memory
	h_x = new int[n];
	//(int*)malloc(n*sizeof(int))
	h_root = new int();
	//(int*)malloc(sizeof(int));
	h_child = new int[2*(n+1)];
	// malloc(2*(n+1)*sizeof(int));
	hipMalloc((void**)&d_root, sizeof(int));
	hipMalloc((void**)&d_x, n*sizeof(int));
	hipMalloc((void**)&d_child, 2*(n+1)*sizeof(int));
	hipMemset(d_child, -1, 2*(n+1)*sizeof(int));


	// fill h_temp and h_x arrays
	for(unsigned int i=0;i<n;i++){
		h_x[i] = i+1;
	}

	for(unsigned int i=0;i<n;i++){
		unsigned int j = random() % (n-i);
		int temp = h_x[i];
		h_x[i] = h_x[i+j];
		h_x[i+j] = temp;
	}
	*h_root = h_x[0];

	// for(unsigned int i=0;i<n;i++){
	// 	std::cout<<h_x[i]<<" ";
	// }
	// std::cout<<"" <<std::endl;


	// copy data to device
	hipMemcpy(d_root, h_root, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_x, h_x, n*sizeof(int), hipMemcpyHostToDevice);


	// kernel call
	dim3 gridSize = 4;
	dim3 blockSize = 4;
	build_binary_tree <<< gridSize, blockSize>>> (d_x, d_child, d_root, n);


	// copy from device back to host
	hipMemcpy(h_child, d_child, 2*(n+1)*sizeof(int), hipMemcpyDeviceToHost);

cout <<  h_child[0] << endl;
	// print tree
	// for( auto i: &h_child){

	// 	std::cout<< i << "\t";
	// };
	std::cout<<"  "<<std::endl;

	// free memory
	delete h_x;
	delete h_root;
	delete h_child;
	hipFree(d_x);
	hipFree(d_root);
	hipFree(d_child);
}